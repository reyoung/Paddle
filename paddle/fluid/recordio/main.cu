#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <chrono>  // NOLINT
#include <cstdio>
#include <ctime>
#include <iostream>
#include <thread>  // NOLINT
#include <vector>

#include "../memory/malloc.h"
#include "../platform/device_context.h"
#include "../platform/place.h"
#include "paddle/fluid/framework/init.h"
#include "paddle/fluid/platform/device_context.h"

typedef std::chrono::high_resolution_clock Clock;

template <class T>
__global__ void im2col(const T *data_im, int num_outs, int im_height,
                       int im_width, int dilation_h, int dilation_w,
                       int filter_height, int filter_width, int stride_height,
                       int stride_width, int padding_height, int padding_width,
                       int col_height, int col_width, T *data_col) {
  const int index =
      (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if (index < num_outs) {
    int w_out = index % col_width;
    int h_out = (index / col_width) % col_height;
    int channel_in = index / col_width / col_height;
    int channel_out = channel_in * filter_height * filter_width;
    int h_in = h_out * stride_height - padding_height;
    int w_in = w_out * stride_width - padding_width;

    data_col += (channel_out * col_height + h_out) * col_width + w_out;
    data_im += (channel_in * im_height + h_in) * im_width + w_in;
    for (int i = 0; i < filter_height; ++i) {
      for (int j = 0; j < filter_width; ++j) {
        int rIdx = h_in + i * dilation_h;
        int cIdx = w_in + j * dilation_w;
        *data_col =
            (rIdx >= im_height || rIdx < 0 || cIdx >= im_width || cIdx < 0)
                ? 0
                : data_im[i * dilation_h * im_width + j * dilation_w];
        data_col += col_height * col_width;
      }
    }
  }
}

void task1(float *im_ptr_d, int im_channels, int im_height, int im_width,
           int filter_height, int filter_width, int col_height, int col_width,
           const std::vector<int> &padding, const std::vector<int> &dilation,
           const std::vector<int> &stride, hipStream_t stream,
           float *col_ptr_d, int dev_id) {
  std::cout << "run..." << std::endl;
  auto t1 = Clock::now();

  typedef std::chrono::high_resolution_clock Clock;
  for (int i = 0; i < 1000; ++i) {
    hipSetDevice(dev_id);
    int num_outputs = im_channels * col_height * col_width;
    int blocks = (num_outputs + 1024 - 1) / 1024;
    int block_x = 512;
    int block_y = (blocks + 512 - 1) / 512;
    dim3 threads(1024, 1);
    dim3 grid(block_x, block_y);

    im2col<float><<<grid, threads, 0, stream>>>(
        im_ptr_d, num_outputs, im_height, im_width, dilation[0], dilation[1],
        filter_height, filter_width, stride[0], stride[1], padding[0],
        padding[1], col_height, col_width, col_ptr_d);
    std::this_thread::sleep_for(std::chrono::microseconds(300));
  }
  hipFree(im_ptr_d);
  hipFree(col_ptr_d);

  auto t2 = Clock::now();
  std::cout
      << "Delta t2-t1: "
      << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count()
      << " milliseconds" << std::endl;
}

int main() {
  paddle::framework::InitDevices(true);
  // Config
  int im_channels = 256 * 2;
  int im_height = 224 * 2;
  int im_width = 224 * 2;
  int filter_height = 5;
  int filter_width = 5;
  std::vector<int> padding{1, 1, 1, 1};
  std::vector<int> dilation{1, 1};
  std::vector<int> stride{2, 2};

  int col_height = (im_height + padding[0] + padding[2] -
                    (dilation[0] * (filter_height - 1) + 1)) /
                       stride[0] +
                   1;
  int col_width = (im_width + padding[1] + padding[3] -
                   (dilation[1] * (filter_width - 1) + 1)) /
                      stride[1] +
                  1;

  size_t im_size = im_channels * im_height * im_width;
  size_t col_size =
      im_channels * filter_height * filter_width * col_height * col_width;

  const int t_cnt = paddle::platform::GetCUDADeviceCount();
  float *src_h;
  std::vector<float *> src_d(t_cnt);
  std::vector<float *> dst_d(t_cnt);
  std::vector<hipStream_t> streams(t_cnt);

  // Init
  src_h = new float[im_size];

  for (int i = 0; i < t_cnt; ++i) {
    src_d[i] = reinterpret_cast<float *>(paddle::memory::Alloc(
        paddle::platform::CUDAPlace(i), im_size * sizeof(float)));
    dst_d[i] = reinterpret_cast<float *>(paddle::memory::Alloc(
        paddle::platform::CUDAPlace(i), col_size * sizeof(float)));
    streams[i] = reinterpret_cast<paddle::platform::CUDADeviceContext *>(
                     paddle::platform::DeviceContextPool::Instance().Get(
                         paddle::platform::CUDAPlace(i)))
                     ->stream();
  }

  for (int i = 0; i < im_size; ++i) {
    src_h[i] = i;
  }

  for (int i = 0; i < t_cnt; ++i) {
    hipSetDevice(i);
    hipMemcpy(src_d[i], src_h, im_size * sizeof(float),
               hipMemcpyHostToDevice);
  }

  // Run
  std::vector<std::thread> workers(t_cnt);
  for (size_t i = 0; i < t_cnt; ++i) {
    workers[i] = std::thread([&, i] {
      task1(src_d[i], im_channels, im_height, im_width, filter_height,
            filter_width, col_height, col_width, padding, dilation, stride,
            streams[i], dst_d[i], i);
    });
  }

  // Wait
  for (auto &worker : workers) {
    worker.join();
    std::cout << "over\n";
  }

  // delete
  delete src_h;
}
